#include "hip/hip_runtime.h"
/*
 * ISPACK FORTRAN SUBROUTINE LIBRARY FOR SCIENTIFIC COMPUTING
 * Copyright (C) 1998--2011 Keiichi Ishioka <ishioka@gfd-dennou.org>
 *
 * This library is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public
 * License as published by the Free Software Foundation; either
 * version 2.1 of the License, or (at your option) any later version.
 *
 * This library is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with this library; if not, write to the Free Software
 * Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA
 * 02110-1301 USA.
 */
#include<hip/hip_runtime.h>
#include<stdio.h>
#define REDUCE(n) if(j<64){sdata[j]+=sdata[j+64];}\
	__syncthreads(); \
	if(j<32){\
	     sdata[j]+=sdata[j+32];\
	     sdata[j]+=sdata[j+16];\
	     sdata[j]+=sdata[j+8];\
	     sdata[j]+=sdata[j+4];\
	     sdata[j]+=sdata[j+2];\
	     sdata[j]+=sdata[j+1];\
	     ws[n]=sdata[0];\
	  }
/*----------------------------------------------------------------*/	  

__global__ void sjwg2s_kernel512(double *p, double *r, double *ws, double *g,
		       int im, int jh, int mm, int nm, int nn, int ipow)
{
   volatile __shared__ double sdata[128];

   int m=blockIdx.x;
   int j=threadIdx.x;
   int n;
   int ns;
   int nsr;   
   int nst;
   int nsrt;   
   double qs10=p[j];
   double qs11=p[128+j];      
   double qs12=p[256+j];                  
   double qs13=p[384+j];                              

   double qs20,qs21,qs22,qs23;
   double qs30,qs31,qs32,qs33;
   double qs40,qs41,qs42,qs43;
   double qs50,qs51,qs52,qs53;
   double qs60,qs61,qs62,qs63;
   double qs70,qs71,qs72,qs73;

   if(m == 0)
     {
	n=m;
	ns=nm;
	qs20=1;
	qs21=1;
	qs22=1;
	qs23=1;
	qs30=qs10;
	qs31=qs11;	
	qs32=qs12;
	qs33=qs13;
	if(ipow == 0)
	  {
	     qs40=(g[im*(jh+j)]+g[im*(jh-j-1)])*p[j+jh];
	     qs41=(g[im*(jh+j+128)]+g[im*(jh-(j+128)-1)])*p[j+128+jh];
	     qs42=(g[im*(jh+j+256)]+g[im*(jh-(j+256)-1)])*p[j+256+jh];
	     qs43=(g[im*(jh+j+384)]+g[im*(jh-(j+384)-1)])*p[j+384+jh];
	     qs50=(g[im*(jh+j)]-g[im*(jh-j-1)])*p[j+jh];
	     qs51=(g[im*(jh+j+128)]-g[im*(jh-(j+128)-1)])*p[j+128+jh];
	     qs52=(g[im*(jh+j+256)]-g[im*(jh-(j+256)-1)])*p[j+256+jh];
	     qs53=(g[im*(jh+j+384)]-g[im*(jh-(j+384)-1)])*p[j+384+jh];
	  }
	else if(ipow == 1)
	  {
	     qs40=(g[im*(jh+j)]+g[im*(jh-j-1)])*p[j+jh]*p[j+jh*3];
	     qs41=(g[im*(jh+j+128)]+g[im*(jh-(j+128)-1)])*p[j+128+jh]*p[j+128+jh*3];
	     qs42=(g[im*(jh+j+256)]+g[im*(jh-(j+256)-1)])*p[j+256+jh]*p[j+256+jh*3];
	     qs43=(g[im*(jh+j+384)]+g[im*(jh-(j+384)-1)])*p[j+384+jh]*p[j+384+jh*3];
	     qs50=(g[im*(jh+j)]-g[im*(jh-j-1)])*p[j+jh]*p[j+jh*3];
	     qs51=(g[im*(jh+j+128)]-g[im*(jh-(j+128)-1)])*p[j+128+jh]*p[j+128+jh*3];
	     qs52=(g[im*(jh+j+256)]-g[im*(jh-(j+256)-1)])*p[j+256+jh]*p[j+256+jh*3];
	     qs53=(g[im*(jh+j+384)]-g[im*(jh-(j+384)-1)])*p[j+384+jh]*p[j+384+jh*3];
	  }
	else if(ipow == 2)
	  {
	     qs40=(g[im*(jh+j)]+g[im*(jh-j-1)])*p[j+jh]*p[j+jh*3]*p[j+jh*3];
	     qs41=(g[im*(jh+j+128)]+g[im*(jh-(j+128)-1)])*p[j+128+jh]*p[j+128+jh*3]*p[j+128+jh*3];
	     qs42=(g[im*(jh+j+256)]+g[im*(jh-(j+256)-1)])*p[j+256+jh]*p[j+256+jh*3]*p[j+256+jh*3];
	     qs43=(g[im*(jh+j+384)]+g[im*(jh-(j+384)-1)])*p[j+384+jh]*p[j+384+jh*3]*p[j+384+jh*3];
	     qs50=(g[im*(jh+j)]-g[im*(jh-j-1)])*p[j+jh]*p[j+jh*3]*p[j+jh*3];
	     qs51=(g[im*(jh+j+128)]-g[im*(jh-(j+128)-1)])*p[j+128+jh]*p[j+128+jh*3]*p[j+128+jh*3];
	     qs52=(g[im*(jh+j+256)]-g[im*(jh-(j+256)-1)])*p[j+256+jh]*p[j+256+jh*3]*p[j+256+jh*3];
	     qs53=(g[im*(jh+j+384)]-g[im*(jh-(j+384)-1)])*p[j+384+jh]*p[j+384+jh*3]*p[j+384+jh*3];
	  }
	__syncthreads();
	sdata[j]=qs40+qs41+qs42+qs43;
	__syncthreads();
	REDUCE(n);
   
	for(n=m+1;n<nn;n=n+2)
	  {
	     __syncthreads();		     
	     sdata[j]=qs50*qs30+qs51*qs31+qs52*qs32+qs53*qs33;
	     __syncthreads();
	     REDUCE(n);
	     qs20=qs20+r[ns+n-1]*qs10*qs30;
	     qs21=qs21+r[ns+n-1]*qs11*qs31;	     
	     qs22=qs22+r[ns+n-1]*qs12*qs32;
	     qs23=qs23+r[ns+n-1]*qs13*qs33;	     
	     
	     __syncthreads();		     	     
	     sdata[j]=qs40*qs20+qs41*qs21+qs42*qs22+qs43*qs23;
	     __syncthreads();
	     REDUCE(n+1);	     
	     
	     qs30=qs30+r[ns+n]*qs10*qs20;
	     qs31=qs31+r[ns+n]*qs11*qs21;	     
	     qs32=qs32+r[ns+n]*qs12*qs22;
	     qs33=qs33+r[ns+n]*qs13*qs23;	     
	     
	     
	  }
	if(((nn-m) % 2) == 1)
	  {
	     n=nn;
	     __syncthreads();		     	     	     
	     sdata[j]=qs50*qs30+qs51*qs31+qs52*qs32+qs53*qs33;
	     __syncthreads();
	     REDUCE(n);
	  }
     }
   else
     {
	ns=nn+2+(m-1)*(2*nn+2-m);
        nsr=(2*nm-m)*m+nm-m;
	qs20=p[j+(4+m-1)*jh];
	qs21=p[128+j+(4+m-1)*jh];      
	qs22=p[256+j+(4+m-1)*jh];                  
	qs23=p[384+j+(4+m-1)*jh];                              
	qs30=qs10*qs20;
	qs31=qs11*qs21;	
	qs32=qs12*qs22;
	qs33=qs13*qs23;
	if(ipow == 0)
	  {
	     qs40=(g[im*(jh+j)+2*m]+g[im*(jh-j-1)+2*m])*p[j+jh];
	     qs41=(g[im*(jh+j+128)+2*m]+g[im*(jh-(j+128)-1)+2*m])*p[j+128+jh];
	     qs42=(g[im*(jh+j+256)+2*m]+g[im*(jh-(j+256)-1)+2*m])*p[j+256+jh];
	     qs43=(g[im*(jh+j+384)+2*m]+g[im*(jh-(j+384)-1)+2*m])*p[j+384+jh];
	     qs60=(g[im*(jh+j)+2*m]-g[im*(jh-j-1)+2*m])*p[j+jh];
	     qs61=(g[im*(jh+j+128)+2*m]-g[im*(jh-(j+128)-1)+2*m])*p[j+128+jh];
	     qs62=(g[im*(jh+j+256)+2*m]-g[im*(jh-(j+256)-1)+2*m])*p[j+256+jh];
	     qs63=(g[im*(jh+j+384)+2*m]-g[im*(jh-(j+384)-1)+2*m])*p[j+384+jh];
	     qs50=(g[im*(jh+j)+2*m+1]+g[im*(jh-j-1)+2*m+1])*p[j+jh];
	     qs51=(g[im*(jh+j+128)+2*m+1]+g[im*(jh-(j+128)-1)+2*m+1])*p[j+128+jh];
	     qs52=(g[im*(jh+j+256)+2*m+1]+g[im*(jh-(j+256)-1)+2*m+1])*p[j+256+jh];
	     qs53=(g[im*(jh+j+384)+2*m+1]+g[im*(jh-(j+384)-1)+2*m+1])*p[j+384+jh];
	     qs70=(g[im*(jh+j)+2*m+1]-g[im*(jh-j-1)+2*m+1])*p[j+jh];
	     qs71=(g[im*(jh+j+128)+2*m+1]-g[im*(jh-(j+128)-1)+2*m+1])*p[j+128+jh];
	     qs72=(g[im*(jh+j+256)+2*m+1]-g[im*(jh-(j+256)-1)+2*m+1])*p[j+256+jh];
	     qs73=(g[im*(jh+j+384)+2*m+1]-g[im*(jh-(j+384)-1)+2*m+1])*p[j+384+jh];
	  }
	else if(ipow == 1)
	  {
	     qs40=(g[im*(jh+j)+2*m]+g[im*(jh-j-1)+2*m])*p[j+jh]*p[j+jh*3];
	     qs41=(g[im*(jh+j+128)+2*m]+g[im*(jh-(j+128)-1)+2*m])*p[j+128+jh]*p[j+128+jh*3];
	     qs42=(g[im*(jh+j+256)+2*m]+g[im*(jh-(j+256)-1)+2*m])*p[j+256+jh]*p[j+256+jh*3];
	     qs43=(g[im*(jh+j+384)+2*m]+g[im*(jh-(j+384)-1)+2*m])*p[j+384+jh]*p[j+384+jh*3];
	     qs60=(g[im*(jh+j)+2*m]-g[im*(jh-j-1)+2*m])*p[j+jh]*p[j+jh*3];
	     qs61=(g[im*(jh+j+128)+2*m]-g[im*(jh-(j+128)-1)+2*m])*p[j+128+jh]*p[j+128+jh*3];
	     qs62=(g[im*(jh+j+256)+2*m]-g[im*(jh-(j+256)-1)+2*m])*p[j+256+jh]*p[j+256+jh*3];
	     qs63=(g[im*(jh+j+384)+2*m]-g[im*(jh-(j+384)-1)+2*m])*p[j+384+jh]*p[j+384+jh*3];
	     qs50=(g[im*(jh+j)+2*m+1]+g[im*(jh-j-1)+2*m+1])*p[j+jh]*p[j+jh*3];
	     qs51=(g[im*(jh+j+128)+2*m+1]+g[im*(jh-(j+128)-1)+2*m+1])*p[j+128+jh]*p[j+128+jh*3];
	     qs52=(g[im*(jh+j+256)+2*m+1]+g[im*(jh-(j+256)-1)+2*m+1])*p[j+256+jh]*p[j+256+jh*3];
	     qs53=(g[im*(jh+j+384)+2*m+1]+g[im*(jh-(j+384)-1)+2*m+1])*p[j+384+jh]*p[j+384+jh*3];
	     qs70=(g[im*(jh+j)+2*m+1]-g[im*(jh-j-1)+2*m+1])*p[j+jh]*p[j+jh*3];
	     qs71=(g[im*(jh+j+128)+2*m+1]-g[im*(jh-(j+128)-1)+2*m+1])*p[j+128+jh]*p[j+128+jh*3];
	     qs72=(g[im*(jh+j+256)+2*m+1]-g[im*(jh-(j+256)-1)+2*m+1])*p[j+256+jh]*p[j+256+jh*3];
	     qs73=(g[im*(jh+j+384)+2*m+1]-g[im*(jh-(j+384)-1)+2*m+1])*p[j+384+jh]*p[j+384+jh*3];
	  }
	else if(ipow == 2)
	  {
	     qs40=(g[im*(jh+j)+2*m]+g[im*(jh-j-1)+2*m])*p[j+jh]*p[j+jh*3]*p[j+jh*3];
	     qs41=(g[im*(jh+j+128)+2*m]+g[im*(jh-(j+128)-1)+2*m])*p[j+128+jh]*p[j+128+jh*3]*p[j+128+jh*3];
	     qs42=(g[im*(jh+j+256)+2*m]+g[im*(jh-(j+256)-1)+2*m])*p[j+256+jh]*p[j+256+jh*3]*p[j+256+jh*3];
	     qs43=(g[im*(jh+j+384)+2*m]+g[im*(jh-(j+384)-1)+2*m])*p[j+384+jh]*p[j+384+jh*3]*p[j+384+jh*3];
	     qs60=(g[im*(jh+j)+2*m]-g[im*(jh-j-1)+2*m])*p[j+jh]*p[j+jh*3]*p[j+jh*3];
	     qs61=(g[im*(jh+j+128)+2*m]-g[im*(jh-(j+128)-1)+2*m])*p[j+128+jh]*p[j+128+jh*3]*p[j+128+jh*3];
	     qs62=(g[im*(jh+j+256)+2*m]-g[im*(jh-(j+256)-1)+2*m])*p[j+256+jh]*p[j+256+jh*3]*p[j+256+jh*3];
	     qs63=(g[im*(jh+j+384)+2*m]-g[im*(jh-(j+384)-1)+2*m])*p[j+384+jh]*p[j+384+jh*3]*p[j+384+jh*3];
	     qs50=(g[im*(jh+j)+2*m+1]+g[im*(jh-j-1)+2*m+1])*p[j+jh]*p[j+jh*3]*p[j+jh*3];
	     qs51=(g[im*(jh+j+128)+2*m+1]+g[im*(jh-(j+128)-1)+2*m+1])*p[j+128+jh]*p[j+128+jh*3]*p[j+128+jh*3];
	     qs52=(g[im*(jh+j+256)+2*m+1]+g[im*(jh-(j+256)-1)+2*m+1])*p[j+256+jh]*p[j+256+jh*3]*p[j+256+jh*3];
	     qs53=(g[im*(jh+j+384)+2*m+1]+g[im*(jh-(j+384)-1)+2*m+1])*p[j+384+jh]*p[j+384+jh*3]*p[j+384+jh*3];
	     qs70=(g[im*(jh+j)+2*m+1]-g[im*(jh-j-1)+2*m+1])*p[j+jh]*p[j+jh*3]*p[j+jh*3];
	     qs71=(g[im*(jh+j+128)+2*m+1]-g[im*(jh-(j+128)-1)+2*m+1])*p[j+128+jh]*p[j+128+jh*3]*p[j+128+jh*3];
	     qs72=(g[im*(jh+j+256)+2*m+1]-g[im*(jh-(j+256)-1)+2*m+1])*p[j+256+jh]*p[j+256+jh*3]*p[j+256+jh*3];
	     qs73=(g[im*(jh+j+384)+2*m+1]-g[im*(jh-(j+384)-1)+2*m+1])*p[j+384+jh]*p[j+384+jh*3]*p[j+384+jh*3];
	  }
	__syncthreads();		     	     	     	
	sdata[j]=qs40*qs20+qs41*qs21+qs42*qs22+qs43*qs23;
	__syncthreads();
	REDUCE(ns-1);
	__syncthreads();		     	     	     	
	sdata[j]=qs50*qs20+qs51*qs21+qs52*qs22+qs53*qs23;
	__syncthreads();
	REDUCE(ns);
	nsrt=nsr;
	nst=ns+1;

	for(n=m+1;n<nn;n=n+2)
	  {
	     qs20=qs20+r[nsrt]*qs10*qs30;
	     qs21=qs21+r[nsrt]*qs11*qs31;	     
	     qs22=qs22+r[nsrt]*qs12*qs32;
	     qs23=qs23+r[nsrt]*qs13*qs33;	     
	     __syncthreads();		     	     	     		     
	     sdata[j]=qs60*qs30+qs61*qs31+qs62*qs32+qs63*qs33;
	     __syncthreads();
	     REDUCE(nst);
	     __syncthreads();
	     sdata[j]=qs70*qs30+qs71*qs31+qs72*qs32+qs73*qs33;
	     __syncthreads();
	     REDUCE(nst+1);
	     __syncthreads();
	     sdata[j]=qs40*qs20+qs41*qs21+qs42*qs22+qs43*qs23;
	     __syncthreads();
	     REDUCE(nst+2);
	     __syncthreads();
	     sdata[j]=qs50*qs20+qs51*qs21+qs52*qs22+qs53*qs23;
	     __syncthreads();
	     REDUCE(nst+3);
	     qs30=qs30+r[nsrt+1]*qs10*qs20;
	     qs31=qs31+r[nsrt+1]*qs11*qs21;	     
	     qs32=qs32+r[nsrt+1]*qs12*qs22;
	     qs33=qs33+r[nsrt+1]*qs13*qs23;	     
	     nsrt=nsrt+2;
	     nst=nst+4; 
	  }
	if(((nn-m) % 2) == 1){
	   n=nn;
	   __syncthreads();	     	   
	   sdata[j]=qs60*qs30+qs61*qs31+qs62*qs32+qs63*qs33;
	   __syncthreads();
	   REDUCE(ns+(n-m-1)*2+1);
	   __syncthreads();	     	   	   
	   sdata[j]=qs70*qs30+qs71*qs31+qs72*qs32+qs73*qs33;
	   __syncthreads();
	   REDUCE(ns+(n-m-1)*2+2);
	}
     }
}
   
/*----------------------------------------------------------------*/

__global__ void sjwg2s_kernel256(double *p, double *r, double *ws, double *g,
		       int im, int jh, int mm, int nm, int nn, int ipow)
{
   volatile __shared__ double sdata[128];

   int m=blockIdx.x;
   int j=threadIdx.x;
   int n;
   int ns;
   int nsr;   
   int nst;
   int nsrt;   
   double qs10=p[j];
   double qs11=p[128+j];                  

   double qs20,qs21;
   double qs30,qs31;
   double qs40,qs41;
   double qs50,qs51;
   double qs60,qs61;
   double qs70,qs71;

   if(m == 0)
     {
	n=m;
	ns=nm;
	qs20=1;
	qs21=1;
	qs30=qs10;
	qs31=qs11;	
	if(ipow == 0)
	  {
	     qs40=(g[im*(jh+j)]+g[im*(jh-j-1)])*p[j+jh];
	     qs41=(g[im*(jh+j+128)]+g[im*(jh-(j+128)-1)])*p[j+128+jh];
	     qs50=(g[im*(jh+j)]-g[im*(jh-j-1)])*p[j+jh];
	     qs51=(g[im*(jh+j+128)]-g[im*(jh-(j+128)-1)])*p[j+128+jh];
	  }
	else if(ipow == 1)
	  {
	     qs40=(g[im*(jh+j)]+g[im*(jh-j-1)])*p[j+jh]*p[j+jh*3];
	     qs41=(g[im*(jh+j+128)]+g[im*(jh-(j+128)-1)])*p[j+128+jh]*p[j+128+jh*3];
	     qs50=(g[im*(jh+j)]-g[im*(jh-j-1)])*p[j+jh]*p[j+jh*3];
	     qs51=(g[im*(jh+j+128)]-g[im*(jh-(j+128)-1)])*p[j+128+jh]*p[j+128+jh*3];
	  }
	else if(ipow == 2)
	  {
	     qs40=(g[im*(jh+j)]+g[im*(jh-j-1)])*p[j+jh]*p[j+jh*3]*p[j+jh*3];
	     qs41=(g[im*(jh+j+128)]+g[im*(jh-(j+128)-1)])*p[j+128+jh]*p[j+128+jh*3]*p[j+128+jh*3];
	     qs50=(g[im*(jh+j)]-g[im*(jh-j-1)])*p[j+jh]*p[j+jh*3]*p[j+jh*3];
	     qs51=(g[im*(jh+j+128)]-g[im*(jh-(j+128)-1)])*p[j+128+jh]*p[j+128+jh*3]*p[j+128+jh*3];
	  }
	__syncthreads();
	sdata[j]=qs40+qs41;
	__syncthreads();
	REDUCE(n);
   
	for(n=m+1;n<nn;n=n+2)
	  {
	     __syncthreads();		     
	     sdata[j]=qs50*qs30+qs51*qs31;
	     __syncthreads();
	     REDUCE(n);
	     qs20=qs20+r[ns+n-1]*qs10*qs30;
	     qs21=qs21+r[ns+n-1]*qs11*qs31;	     
	     
	     __syncthreads();		     	     
	     sdata[j]=qs40*qs20+qs41*qs21;
	     __syncthreads();
	     REDUCE(n+1);	     
	     
	     qs30=qs30+r[ns+n]*qs10*qs20;
	     qs31=qs31+r[ns+n]*qs11*qs21;	     
	     
	  }
	if(((nn-m) % 2) == 1)
	  {
	     n=nn;
	     __syncthreads();		     	     	     
	     sdata[j]=qs50*qs30+qs51*qs31;
	     __syncthreads();
	     REDUCE(n);
	  }
     }
   else
     {
	ns=nn+2+(m-1)*(2*nn+2-m);
        nsr=(2*nm-m)*m+nm-m;
	qs20=p[j+(4+m-1)*jh];
	qs21=p[128+j+(4+m-1)*jh];                  
	qs30=qs10*qs20;
	qs31=qs11*qs21;	
	if(ipow == 0)
	  {
	     qs40=(g[im*(jh+j)+2*m]+g[im*(jh-j-1)+2*m])*p[j+jh];
	     qs41=(g[im*(jh+j+128)+2*m]+g[im*(jh-(j+128)-1)+2*m])*p[j+128+jh];
	     qs60=(g[im*(jh+j)+2*m]-g[im*(jh-j-1)+2*m])*p[j+jh];
	     qs61=(g[im*(jh+j+128)+2*m]-g[im*(jh-(j+128)-1)+2*m])*p[j+128+jh];
	     qs50=(g[im*(jh+j)+2*m+1]+g[im*(jh-j-1)+2*m+1])*p[j+jh];
	     qs51=(g[im*(jh+j+128)+2*m+1]+g[im*(jh-(j+128)-1)+2*m+1])*p[j+128+jh];
	     qs70=(g[im*(jh+j)+2*m+1]-g[im*(jh-j-1)+2*m+1])*p[j+jh];
	     qs71=(g[im*(jh+j+128)+2*m+1]-g[im*(jh-(j+128)-1)+2*m+1])*p[j+128+jh];
	  }
	else if(ipow == 1)
	  {
	     qs40=(g[im*(jh+j)+2*m]+g[im*(jh-j-1)+2*m])*p[j+jh]*p[j+jh*3];
	     qs41=(g[im*(jh+j+128)+2*m]+g[im*(jh-(j+128)-1)+2*m])*p[j+128+jh]*p[j+128+jh*3];
	     qs60=(g[im*(jh+j)+2*m]-g[im*(jh-j-1)+2*m])*p[j+jh]*p[j+jh*3];
	     qs61=(g[im*(jh+j+128)+2*m]-g[im*(jh-(j+128)-1)+2*m])*p[j+128+jh]*p[j+128+jh*3];
	     qs50=(g[im*(jh+j)+2*m+1]+g[im*(jh-j-1)+2*m+1])*p[j+jh]*p[j+jh*3];
	     qs51=(g[im*(jh+j+128)+2*m+1]+g[im*(jh-(j+128)-1)+2*m+1])*p[j+128+jh]*p[j+128+jh*3];
	     qs70=(g[im*(jh+j)+2*m+1]-g[im*(jh-j-1)+2*m+1])*p[j+jh]*p[j+jh*3];
	     qs71=(g[im*(jh+j+128)+2*m+1]-g[im*(jh-(j+128)-1)+2*m+1])*p[j+128+jh]*p[j+128+jh*3];
	  }
	else if(ipow == 2)
	  {
	     qs40=(g[im*(jh+j)+2*m]+g[im*(jh-j-1)+2*m])*p[j+jh]*p[j+jh*3]*p[j+jh*3];
	     qs41=(g[im*(jh+j+128)+2*m]+g[im*(jh-(j+128)-1)+2*m])*p[j+128+jh]*p[j+128+jh*3]*p[j+128+jh*3];
	     qs60=(g[im*(jh+j)+2*m]-g[im*(jh-j-1)+2*m])*p[j+jh]*p[j+jh*3]*p[j+jh*3];
	     qs61=(g[im*(jh+j+128)+2*m]-g[im*(jh-(j+128)-1)+2*m])*p[j+128+jh]*p[j+128+jh*3]*p[j+128+jh*3];
	     qs50=(g[im*(jh+j)+2*m+1]+g[im*(jh-j-1)+2*m+1])*p[j+jh]*p[j+jh*3]*p[j+jh*3];
	     qs51=(g[im*(jh+j+128)+2*m+1]+g[im*(jh-(j+128)-1)+2*m+1])*p[j+128+jh]*p[j+128+jh*3]*p[j+128+jh*3];
	     qs70=(g[im*(jh+j)+2*m+1]-g[im*(jh-j-1)+2*m+1])*p[j+jh]*p[j+jh*3]*p[j+jh*3];
	     qs71=(g[im*(jh+j+128)+2*m+1]-g[im*(jh-(j+128)-1)+2*m+1])*p[j+128+jh]*p[j+128+jh*3]*p[j+128+jh*3];
	  }
	__syncthreads();
	sdata[j]=qs40*qs20+qs41*qs21;
	__syncthreads();
	REDUCE(ns-1);
	__syncthreads();		     	     	     	
	sdata[j]=qs50*qs20+qs51*qs21;
	__syncthreads();
	REDUCE(ns);
	nsrt=nsr;
	nst=ns+1;

	for(n=m+1;n<nn;n=n+2)
	  {
	     qs20=qs20+r[nsrt]*qs10*qs30;
	     qs21=qs21+r[nsrt]*qs11*qs31;	     
	     __syncthreads();		     	     	     		     
	     sdata[j]=qs60*qs30+qs61*qs31;
	     __syncthreads();
	     REDUCE(nst);
	     __syncthreads();
	     sdata[j]=qs70*qs30+qs71*qs31;
	     __syncthreads();
	     REDUCE(nst+1);
	     __syncthreads();
	     sdata[j]=qs40*qs20+qs41*qs21;
	     __syncthreads();
	     REDUCE(nst+2);
	     __syncthreads();
	     sdata[j]=qs50*qs20+qs51*qs21;
	     __syncthreads();
	     REDUCE(nst+3);
	     qs30=qs30+r[nsrt+1]*qs10*qs20;
	     qs31=qs31+r[nsrt+1]*qs11*qs21;	     
	     nsrt=nsrt+2;
	     nst=nst+4; 
	  }
	if(((nn-m) % 2) == 1){
	   n=nn;
	   __syncthreads();	     	   
	   sdata[j]=qs60*qs30+qs61*qs31;
	   __syncthreads();
	   REDUCE(ns+(n-m-1)*2+1);
	   __syncthreads();	     	   	   
	   sdata[j]=qs70*qs30+qs71*qs31;
	   __syncthreads();
	   REDUCE(ns+(n-m-1)*2+2);
	}
     }
}
   
/*----------------------------------------------------------------*/

__global__ void sjwg2s_kernel128(double *p, double *r, double *ws, double *g,
		       int im, int jh, int mm, int nm, int nn, int ipow)
{
   volatile __shared__ double sdata[128];

   int m=blockIdx.x;
   int j=threadIdx.x;
   int n;
   int ns;
   int nsr;   
   int nst;
   int nsrt;   
   double qs10=p[j];

   double qs20;
   double qs30;
   double qs40;
   double qs50;
   double qs60;
   double qs70;

   if(m == 0)
     {
	n=m;
	ns=nm;
	qs20=1;
	qs30=qs10;
	if(ipow == 0)
	  {
	     qs40=(g[im*(jh+j)]+g[im*(jh-j-1)])*p[j+jh];
	     qs50=(g[im*(jh+j)]-g[im*(jh-j-1)])*p[j+jh];
	  }
	else if(ipow == 1)
	  {
	     qs40=(g[im*(jh+j)]+g[im*(jh-j-1)])*p[j+jh]*p[j+jh*3];
	     qs50=(g[im*(jh+j)]-g[im*(jh-j-1)])*p[j+jh]*p[j+jh*3];
	  }
	else if(ipow == 2)
	  {
	     qs40=(g[im*(jh+j)]+g[im*(jh-j-1)])*p[j+jh]*p[j+jh*3]*p[j+jh*3];
	     qs50=(g[im*(jh+j)]-g[im*(jh-j-1)])*p[j+jh]*p[j+jh*3]*p[j+jh*3];
	  }
	__syncthreads();
	sdata[j]=qs40;
	__syncthreads();
	REDUCE(n);
   
	for(n=m+1;n<nn;n=n+2)
	  {
	     __syncthreads();		     
	     sdata[j]=qs50*qs30;
	     __syncthreads();
	     REDUCE(n);
	     qs20=qs20+r[ns+n-1]*qs10*qs30;
	     
	     __syncthreads();		     	     
	     sdata[j]=qs40*qs20;
	     __syncthreads();
	     REDUCE(n+1);	     
	     
	     qs30=qs30+r[ns+n]*qs10*qs20;
	     
	  }
	if(((nn-m) % 2) == 1)
	  {
	     n=nn;
	     __syncthreads();		     	     	     
	     sdata[j]=qs50*qs30;
	     __syncthreads();
	     REDUCE(n);
	  }
     }
   else
     {
	ns=nn+2+(m-1)*(2*nn+2-m);
        nsr=(2*nm-m)*m+nm-m;
	qs20=p[j+(4+m-1)*jh];
	qs30=qs10*qs20;
	if(ipow == 0)
	  {
	     qs40=(g[im*(jh+j)+2*m]+g[im*(jh-j-1)+2*m])*p[j+jh];
	     qs60=(g[im*(jh+j)+2*m]-g[im*(jh-j-1)+2*m])*p[j+jh];
	     qs50=(g[im*(jh+j)+2*m+1]+g[im*(jh-j-1)+2*m+1])*p[j+jh];
	     qs70=(g[im*(jh+j)+2*m+1]-g[im*(jh-j-1)+2*m+1])*p[j+jh];
	  }
	else if(ipow == 1)
	  {
	     qs40=(g[im*(jh+j)+2*m]+g[im*(jh-j-1)+2*m])*p[j+jh]*p[j+jh*3];
	     qs60=(g[im*(jh+j)+2*m]-g[im*(jh-j-1)+2*m])*p[j+jh]*p[j+jh*3];
	     qs50=(g[im*(jh+j)+2*m+1]+g[im*(jh-j-1)+2*m+1])*p[j+jh]*p[j+jh*3];
	     qs70=(g[im*(jh+j)+2*m+1]-g[im*(jh-j-1)+2*m+1])*p[j+jh]*p[j+jh*3];
	  }
	else if(ipow == 2)
	  {
	     qs40=(g[im*(jh+j)+2*m]+g[im*(jh-j-1)+2*m])*p[j+jh]*p[j+jh*3]*p[j+jh*3];
	     qs60=(g[im*(jh+j)+2*m]-g[im*(jh-j-1)+2*m])*p[j+jh]*p[j+jh*3]*p[j+jh*3];
	     qs50=(g[im*(jh+j)+2*m+1]+g[im*(jh-j-1)+2*m+1])*p[j+jh]*p[j+jh*3]*p[j+jh*3];
	     qs70=(g[im*(jh+j)+2*m+1]-g[im*(jh-j-1)+2*m+1])*p[j+jh]*p[j+jh*3]*p[j+jh*3];
	  }
	__syncthreads();
	sdata[j]=qs40*qs20;
	__syncthreads();
	REDUCE(ns-1);
	__syncthreads();		     	     	     	
	sdata[j]=qs50*qs20;
	__syncthreads();
	REDUCE(ns);
	nsrt=nsr;
	nst=ns+1;

	for(n=m+1;n<nn;n=n+2)
	  {
	     qs20=qs20+r[nsrt]*qs10*qs30;
	     __syncthreads();		     	     	     		     
	     sdata[j]=qs60*qs30;
	     __syncthreads();
	     REDUCE(nst);
	     __syncthreads();
	     sdata[j]=qs70*qs30;
	     __syncthreads();
	     REDUCE(nst+1);
	     __syncthreads();
	     sdata[j]=qs40*qs20;
	     __syncthreads();
	     REDUCE(nst+2);
	     __syncthreads();
	     sdata[j]=qs50*qs20;
	     __syncthreads();
	     REDUCE(nst+3);
	     qs30=qs30+r[nsrt+1]*qs10*qs20;
	     nsrt=nsrt+2;
	     nst=nst+4; 
	  }
	if(((nn-m) % 2) == 1){
	   n=nn;
	   __syncthreads();	     	   
	   sdata[j]=qs60*qs30;
	   __syncthreads();
	   REDUCE(ns+(n-m-1)*2+1);
	   __syncthreads();	     	   	   
	   sdata[j]=qs70*qs30;
	   __syncthreads();
	   REDUCE(ns+(n-m-1)*2+2);
	}
     }
}
   
/*----------------------------------------------------------------*/

extern "C" void sjwg2s_(int *mm, int *nm, int *nn, int *im, int *jm, 
			double *p, double *r, double *ws, double *g,
			int *ipow, long *ip)
{
   int jh = *jm/2;
   size_t sizews,sizeg;
   double *pd;
   double *rd;
   double *wsd;         
   double *gd;

   pd=(double *) ip[0];
   rd=(double *) ip[1];
   wsd=(double *) ip[2];
   gd=(double *) ip[3];

   sizeg=sizeof(double)*(*jm)*(*im);
   sizews=sizeof(double)*((2*(*nn)+1-*mm)*(*mm)+(*nn)+1);

   hipMemcpy(gd,g,sizeg,hipMemcpyHostToDevice);
   
   if(jh == 512)
     {
	sjwg2s_kernel512<<<(*mm+1),128>>>(pd,rd,wsd,gd,*im,jh,*mm,*nm,*nn,*ipow);
     }
   else if(jh == 256)
     {
	sjwg2s_kernel256<<<(*mm+1),128>>>(pd,rd,wsd,gd,*im,jh,*mm,*nm,*nn,*ipow);
     }
   else if(jh == 128)
     {
	sjwg2s_kernel128<<<(*mm+1),128>>>(pd,rd,wsd,gd,*im,jh,*mm,*nm,*nn,*ipow);
     }
   else
     {
        hipFree(pd);
        hipFree(rd);   
	hipFree(wsd);
	hipFree(gd);
	puts("***** ERROR (SJWG2S) ***  This value of JM is not supported.");	
        exit(1);
     }
   
   hipMemcpy(ws,wsd,sizews,hipMemcpyDeviceToHost);
   
   hipDeviceSynchronize(); 
}

/*----------------------------------------------------------------*/

__global__ void sjws2g_kernel(double *p, double *r, double *ws, double *g,
		       int im, int jh, int mm, int nm, int nn, int ipow)
{
   int m=blockIdx.x;
   int j=threadIdx.x;
   int i;
   int n;
   int ns;
   int nsr;   
   int nst;
   int nsrt;   
   double qs1=p[j];
   double qs2;   
   double qs3;      
   double qs4;         
   double qs5;            
   double qs6=0;   
   double qs7=0;

   if(m == 0)
     {
	n=m;
	ns=nm;
	qs2=1;
	qs3=qs1;
	qs4=ws[n];
	qs5=0;

	for(n=m+1;n<nn;n=n+2)
	  {
	     qs5=qs5+ws[n]*qs3;
	     qs2=qs2+r[ns+n-1]*qs1*qs3;
	     qs4=qs4+ws[n+1]*qs2;
	     qs3=qs3+r[ns+n]*qs1*qs2;
	  }
	if(((nn-m) % 2) == 1){
	   n=nn;
	   qs5=qs5+ws[n]*qs3;
	}
	if(ipow == 0)
	  {
	     g[im*(jh+j)]  =(qs4+qs5);
	     g[im*(jh-j-1)]=(qs4-qs5);
	  }
	else if(ipow == 1)
	  {
	     g[im*(jh+j)]  =(qs4+qs5)*p[jh*3+j];
	     g[im*(jh-j-1)]=(qs4-qs5)*p[jh*3+j];
	  }
	else if(ipow == 2)
	  {
	     g[im*(jh+j)]  =(qs4+qs5)*p[jh*3+j]*p[jh*3+j];
	     g[im*(jh-j-1)]=(qs4-qs5)*p[jh*3+j]*p[jh*3+j];
	  }
	
	g[im*j+1]     =0;     
	g[im*(jh+j)+1]=0;	
	for(i=2*(mm+1);i<im;i++)
	  {
	     g[im*j+i]     =0;
	     g[im*(jh+j)+i]=0;
	  }
     }
   else
     {
	ns=nn+2+(m-1)*(2*nn+2-m);
        nsr=(2*nm-m)*m+nm-m;
	qs2=p[j+(4+m-1)*jh];
	qs3=qs1*qs2;
	qs4=ws[ns-1]*qs2;
	qs5=ws[ns  ]*qs2;	
	qs6=0;
	qs7=0;
	
	nsrt=nsr;
	nst=ns+1;

	for(n=m+1;n<nn;n=n+2)
	  {
	     qs2=qs2+r[nsrt]*qs1*qs3;	     
	     qs6=qs6+ws[nst]*qs3;
	     qs7=qs7+ws[nst+1]*qs3;
	     qs4=qs4+ws[nst+2]*qs2;
	     qs5=qs5+ws[nst+3]*qs2;
	     qs3=qs3+r[nsrt+1]*qs1*qs2;	     	     
	     nsrt=nsrt+2;
	     nst=nst+4; 
	  }
	if(((nn-m) % 2) == 1){
	   n=nn;
	   qs6=qs6+ws[ns+(n-m-1)*2+1]*qs3;
	   qs7=qs7+ws[ns+(n-m-1)*2+2]*qs3; 
	}
	if(ipow == 0)
	  {
	     g[im*(jh+j)+2*m  ]  =(qs4+qs6);
	     g[im*(jh+j)+2*m+1]  =(qs5+qs7);
	     g[im*(jh-j-1)+2*m]  =(qs4-qs6);
	     g[im*(jh-j-1)+2*m+1]=(qs5-qs7);
	  }
	else if(ipow == 1)
	  {
	     g[im*(jh+j)+2*m  ]  =(qs4+qs6)*p[jh*3+j];
	     g[im*(jh+j)+2*m+1]  =(qs5+qs7)*p[jh*3+j];
	     g[im*(jh-j-1)+2*m]  =(qs4-qs6)*p[jh*3+j];
	     g[im*(jh-j-1)+2*m+1]=(qs5-qs7)*p[jh*3+j];
	  }
	else if(ipow == 2)
	  {
	     g[im*(jh+j)+2*m  ]  =(qs4+qs6)*p[jh*3+j]*p[jh*3+j];
	     g[im*(jh+j)+2*m+1]  =(qs5+qs7)*p[jh*3+j]*p[jh*3+j];
	     g[im*(jh-j-1)+2*m]  =(qs4-qs6)*p[jh*3+j]*p[jh*3+j];
	     g[im*(jh-j-1)+2*m+1]=(qs5-qs7)*p[jh*3+j]*p[jh*3+j];
	  }
     }
}
   
/*----------------------------------------------------------------*/

extern "C" void sjws2g_(int *mm, int *nm, int *nn, int *im, int *jm, 
			double *p, double *r, double *ws, double *g,
			int *ipow, long *ip)
{
   int jh = *jm/2;
   size_t sizews,sizeg;
   double *pd;
   double *rd;
   double *wsd;         
   double *gd;

   pd=(double *) ip[0];
   rd=(double *) ip[1];
   wsd=(double *) ip[2];
   gd=(double *) ip[3];

   sizeg=sizeof(double)*(*jm)*(*im);
   sizews=sizeof(double)*((2*(*nn)+1-*mm)*(*mm)+(*nn)+1);

   hipMemcpy(wsd,ws,sizews,hipMemcpyHostToDevice); 
   
   sjws2g_kernel<<<(*mm+1),jh>>>(pd,rd,wsd,gd,*im,jh,*mm,*nm,*nn,*ipow);

   hipMemcpy(g,gd,sizeg,hipMemcpyDeviceToHost);
   
   hipDeviceSynchronize(); 
}

/*----------------------------------------------------------------*/

extern "C" void sjvopn_(int *mm, int *nm, int *jm, int *im, 
			double *p, double *r, long *ip)
{
   int jh = *jm/2;
   size_t sizep,sizer,sizews,sizeg;
   double *pd;
   double *rd;
   double *wsd;
   double *gd;   

   sizep=sizeof(double)*jh*(*mm+4);
   sizer=sizeof(double)*((*mm+1)*(2*(*nm)-(*mm)-1)+1);
   sizews=sizeof(double)*((2*(*nm)+1-*mm)*(*mm)+(*nm)+1);
   sizeg=sizeof(double)*(*jm)*(*im);

   hipMalloc((void **)&pd,sizep);
   hipMalloc((void **)&rd,sizer);
   hipMalloc((void **)&wsd,sizews);
   hipMalloc((void **)&gd,sizeg);
   
   hipMemcpy(pd,p,sizep,hipMemcpyHostToDevice);
   hipMemcpy(rd,r,sizer,hipMemcpyHostToDevice);
   
   ip[0]=(long)pd;
   ip[1]=(long)rd;   
   ip[2]=(long)wsd;
   ip[3]=(long)gd;
}

/*----------------------------------------------------------------*/
   
extern "C" void sjvcls_(long *ip)
{
   double *pd;
   double *rd;
   double *wsd;         
   double *gd;   
   
   pd=(double *) ip[0];
   rd=(double *) ip[1];
   wsd=(double *) ip[2];
   gd=(double *) ip[3];

   hipFree(pd);
   hipFree(rd);   
   hipFree(wsd);
   hipFree(gd);
}
   
